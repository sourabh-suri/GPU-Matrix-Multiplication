#define BlockDim 16
#define TileWidth 16


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//*********************************************************************
//Function to multiply two matrices using GPU with tiling optimisation

__global__ void tiling_mul(float *gpu_a, float *gpu_b, float *gpu_result,int N) 
{
    // Considering Tile width same as BloackDim
    __shared__ float tile_a[TileWidth][TileWidth];
    __shared__ float tile_b[TileWidth][TileWidth];
    
    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    
   // Identify the row and column of the C element to work on
    int row = by * TileWidth + ty;
    int col = bx * TileWidth + tx;
    float temp = 0;
    int idx,idy;
    
    // Loop over the A and B tiles required to compute the C element
    // Built-in variable gridDim specifies the size (or dimension) of the grid. 
    // Here m iterator defines the sub-block in action
    for (int m = 0; m < gridDim.x; ++m) 
    {
        idx = row * N + m * BlockDim + tx;
        idy = (m * BlockDim + ty) * N + col;
        // Check if N is divisible by BlockDim
        tile_a[ty][tx] = (idx < N*N) ? gpu_a[idx]: 0;
        tile_b[ty][tx] = (idy < N*N) ? gpu_b[idy]: 0;

        for (int k = 0; k < BlockDim; ++k) 
        {
            temp  = temp + tile_a[ty][k] * tile_b[k][tx];
        }
    }
    if(row < N && col < N)
    {
        gpu_result[row * N + col] = temp;
    }
}
 


//*********************************************************************
// Main Function
int main(int argc, char *argv[])
{   
int N=4;
    N= atoi(argv[1]);
    // allocate memory in cpu
    float *cpu_a, *cpu_b, *cpu_c;
    cpu_a = (float *)malloc(sizeof(float)*N*N);
    cpu_b = (float *)malloc(sizeof(float)*N*N);
    cpu_c = (float *)malloc(sizeof(float)*N*N);

   
    // Allocate memory on the gpu 
    float *gpu_a, *gpu_b, *gpu_c;
    hipMalloc(&gpu_a, sizeof(float)*N*N);
    hipMalloc(&gpu_b, sizeof(float)*N*N);
    hipMalloc(&gpu_c, sizeof(float)*N*N);
    
    // Initialize mat A and B
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            cpu_a[i * N + j] = (float)rand()/(float)(RAND_MAX); 
            //Range 0 to 1
            cpu_b[i * N + j] = (float)rand()/(float)(RAND_MAX); 
            //Range 0 to 1
        }
    }
    if(N<10){
	//displays Matrix 1 only for N<10
	printf("Matrix 1 is \n");
	for (int i = 0; i < N; i++) {
	for (int j = 0; j < N; j++)
	    printf("%f ", cpu_a[i * N + j]);
	printf("\n");
	}
	//displays Matrix 2 only for N<10
	printf("Matrix 2 is \n");
	for (int i = 0; i < N; i++) {
	for (int j = 0; j < N; j++)
	    printf("%f ", cpu_b[i * N + j]);
	printf("\n");
	}
    } 	


    // copy matrix A and B from cpu to gpu memory
    hipMemcpy(gpu_a, cpu_a, sizeof(float)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, cpu_b, sizeof(float)*N*N, hipMemcpyHostToDevice);
 
    //dim3 type is an integer vector to specify dimensions.
   
    dim3 dimGrid((unsigned int) (N-1)/BlockDim +1,(unsigned int) (N-1)/BlockDim +1,1);
    dim3 dimBlock(BlockDim, BlockDim,1);

    // Kernel
    tiling_mul<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c, N);       

    // Transefr results from gpu to cpu 
    hipMemcpy(cpu_c, gpu_c, sizeof(float)*N*N, hipMemcpyDeviceToHost);
   
   // Print if N<10
	 if (N<10) {
            printf("Done, Matrix Result is\n");
	    for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
		   printf("%f ", cpu_c[i * N + j]);
		}
		printf("\n");
	    }
	 }
    
    // free memory
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    free(cpu_a);
    free(cpu_b);
    free(cpu_c);
}
