#define BlockDim 16


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


//*********************************************************************
//Function to multiply two matrices using GPU without any optimisation
__global__ void mul(float *a,float *b, float *c,int N)
{ 
    float temp = 0;
    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    
   // Identify the row and column of the C element to work on
    int row = by * BlockDim + ty;
    int col = bx * BlockDim + tx;

    if( col < N && row < N) 
    {
        for(int i = 0; i < N; i++) 
        {
        // mat a is row major traversed and mat b is col major traversed
            temp = temp + a[i + row * N] * b[i * N + col]; 
        }
        c[row * N + col] = temp;
    }
} 


//*********************************************************************
// Main Function
int main(int argc, char *argv[])
{   
int N=4;
    N= atoi(argv[1]);
    // allocate memory in cpu
    float *cpu_a, *cpu_b, *cpu_c;
    cpu_a = (float *)malloc(sizeof(float)*N*N);
    cpu_b = (float *)malloc(sizeof(float)*N*N);
    cpu_c = (float *)malloc(sizeof(float)*N*N);

   
    // Allocate memory on the gpu 
    float *gpu_a, *gpu_b, *gpu_c;
    hipMalloc(&gpu_a, sizeof(float)*N*N);
    hipMalloc(&gpu_b, sizeof(float)*N*N);
    hipMalloc(&gpu_c, sizeof(float)*N*N);
    
    // Initialize mat A and B
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            cpu_a[i * N + j] = (float)rand()/(float)(RAND_MAX); 
            //Range 0 to 1
            cpu_b[i * N + j] = (float)rand()/(float)(RAND_MAX); 
            //Range 0 to 1
        }
    }
    if(N<10){
	//displays Matrix 1 only for N<10
	printf("Matrix 1 is \n");
	for (int i = 0; i < N; i++) {
	for (int j = 0; j < N; j++)
	    printf("%f ", cpu_a[i * N + j]);
	printf("\n");
	}
	//displays Matrix 2 only for N<10
	printf("Matrix 2 is \n");
	for (int i = 0; i < N; i++) {
	for (int j = 0; j < N; j++)
	    printf("%f ", cpu_b[i * N + j]);
	printf("\n");
	}
    } 	


    // copy matrix A and B from cpu to gpu memory
    hipMemcpy(gpu_a, cpu_a, sizeof(float)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, cpu_b, sizeof(float)*N*N, hipMemcpyHostToDevice);
 
    //dim3 type is defined integer vector to specify dimensions.
    dim3 dimBlock(BlockDim, BlockDim,1);
    dim3 dimGrid((unsigned int) (N-1)/BlockDim +1,(unsigned int) (N-1)/BlockDim +1,1);

    // Kernel  
    mul<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c, N);   

    // Transefr results from gpu to cpu 
    hipMemcpy(cpu_c, gpu_c, sizeof(float)*N*N, hipMemcpyDeviceToHost);
   
   // Print if N<10
	 if (N<10) {
            printf("Done, Matrix Result is\n");
	    for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
		   printf("%f ", cpu_c[i * N + j]);
		}
		printf("\n");
	    }
	 }
    
    // free memory
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    free(cpu_a);
    free(cpu_b);
    free(cpu_c);
}
